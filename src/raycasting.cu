#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "raycasting.h"

////////////////////////////////////////////////////////////////////////////////
// Helper functions
////////////////////////////////////////////////////////////////////////////////
float Max(float x, float y) {
	return (x > y) ? x : y;
}

float Min(float x, float y) {
	return (x < y) ? x : y;
}

int iDivUp(int a, int b) {
	return ((a % b) != 0) ? (a / b + 1) : (a / b);
}

__device__ float lerpf(float a, float b, float c) {
	return a + (b - a) * c;
}

__device__ float vecLen(float4 a, float4 b) {
	return ((b.x - a.x) * (b.x - a.x) + (b.y - a.y) * (b.y - a.y)
			+ (b.z - a.z) * (b.z - a.z));
}

__device__ TColor make_color(float r, float g, float b, float a) {
	return ((int) (a * 255.0f) << 24) | ((int) (b * 255.0f) << 16)
			| ((int) (g * 255.0f) << 8) | ((int) (r * 255.0f) << 0);
}

////////////////////////////////////////////////////////////////////////////////
// Global data handlers and parameters
////////////////////////////////////////////////////////////////////////////////
//Texture reference and channel descriptor for image texture
texture<uchar4, 2, hipReadModeNormalizedFloat> texImage;
hipChannelFormatDesc uchar4tex = hipCreateChannelDesc<uchar4>();

//CUDA array descriptor
hipArray *a_Src;

#include "utility.cuh"
////////////////////////////////////////////////////////////////////////////////
//Raycasting Classes' functions
////////////////////////////////////////////////////////////////////////////////
CUDA_CALLABLE_MEMBER const Vector2 Vector2::operator*(const float &q) const{
	return (Vector2(this->x * 3, this->y * 3));
}

CUDA_CALLABLE_MEMBER const Vector2 Vector2::operator+(const Vector2& q) const{
	return (Vector2(this->x + q.x, this->y + q.y));
}

CUDA_CALLABLE_MEMBER const Vector2 Vector2::operator-(const Vector2& q) const{
	return (Vector2(x - q.x, y - q.y));
}

CUDA_CALLABLE_MEMBER const Vector2 Vector2::direction() const{
	float length = sqrtf((this->x * this->x) + (this->y * this->y));
	return Vector2(this->x / length, this->y / length);
}

CUDA_CALLABLE_MEMBER const Vector3 Vector3::operator*(const float q) const{
	return (Vector3(this->x * q, this->y * q, this->z * q));
}

CUDA_CALLABLE_MEMBER const Vector3 Vector3::operator+(const Vector3& q) const{
	return (Vector3(this->x + q.x, this->y + q.y, this->z + q.z));
}

CUDA_CALLABLE_MEMBER const Vector3 Vector3::operator-(const Vector3& q) const {
	return (Vector3(this->x - q.x, this->y - q.y, this->z - q.z));
}

CUDA_CALLABLE_MEMBER const Vector3 Vector3::direction() const{
	float length = sqrtf((this->x * this->x) + (this->y * this->y) + (this->z * this->z));
	return Vector3(this->x / length, this->y / length, this->z / length);
}

CUDA_CALLABLE_MEMBER const Vector3& Ray::origin() const {
	return m_origin;
}
CUDA_CALLABLE_MEMBER const Vector3& Ray::direction() const {
	return m_direction;
}

CUDA_CALLABLE_MEMBER const Vector3& Triangle::vertex(int i) const {
	return m_vertex[i];
}
CUDA_CALLABLE_MEMBER const Vector3& Triangle::normal(int i) const {
	return m_normal[i];
}

////////////////////////////////////////////////////////////////////////////////
//Raycasting device functions
////////////////////////////////////////////////////////////////////////////////
__device__ Ray computeEyeRay(float x, float y, int width, int height,
		const Camera& camera) {
	const float aspect = float(height) / width;

	// Compute the side of a square at z = -1 based on our
	// horizontal left-edge-to-right-edge field of view
	const float s = -2.0f * tan(camera.fieldOfViewX * 0.5f);
	const Vector3& start = Vector3((x / width - 0.5f) * s,
			-(y / height - 0.5f) * s * aspect, 1.0f) * camera.zNear;
	return Ray(start, start.direction());
}

__device__ bool sampleRayTriangle(const Scene& scene, int x, int y,
		const Ray& R, const Triangle& T, Radiance3& radiance, float& distance) {
	float weight[3];
	const float d = intersect(R, T, weight);
	if (d >= distance) {
		return false;
	}
	// This intersection is closer than the previous one
	// Intersection point
	const Vector3& P = R.origin() + R.direction() * d;
	// Find the interpolated vertex normal at the intersection
	const Vector3& n = (T.normal(0) * weight[0] + T.normal(1) * weight[1]
			+ T.normal(2) * weight[2]).direction();
	const Vector3& w_o = -R.direction();

	//shade(scene, T, P, n, w_o, radiance);

	// Debugging intersect: set to white on any intersection
	//radiance = Radiance3(1, 1, 1);

	// Debugging barycentric
	//radiance = Radiance3(weight[0], weight[1], weight[2]) / 15;

	return true;
}

__device__ float intersect(const Ray& R, const Triangle& T, float weight[3]) {
	const Vector3& e1 = T.vertex(1) - T.vertex(0);
	const Vector3& e2 = T.vertex(2) - T.vertex(0);
	const Vector3& q = R.direction().cross(e2);

	const float a = e1.dot(q);
	const Vector3& s = R.origin() - T.vertex(0);
	const Vector3& r = s.cross(e1);

	// Barycentric vertex weights
	weight[1] = s.dot(q) / a;
	weight[2] = R.direction().dot(r) / a;
	weight[0] = 1.0f - (weight[1] + weight[2]);
	const float dist = e2.dot(r) / a;
	const float epsilon = 1e-7f;

	const float epsilon2 = 1e-10;

	if ((a <= epsilon) || (weight[0] < -epsilon2) || (weight[1] < -epsilon2)
			|| (weight[2] < -epsilon2) || (dist <= 0.0f)) {
		// The ray is nearly parallel to the triangle, or the
		// intersection lies outside the triangle or behind
		// the ray origin: "infinite" distance until intersection.
		return INFINITY;
	} else {
		return dist;
	}
}

////////////////////////////////////////////////////////////////////////////////
// kernels
////////////////////////////////////////////////////////////////////////////////
__global__ void Clear(TColor *dst, int imageW, int imageH) {
	const int ix = blockDim.x * blockIdx.x + threadIdx.x;
	const int iy = blockDim.y * blockIdx.y + threadIdx.y;

	if (ix < imageW && iy < imageH) {
		dst[imageW * iy + ix] = make_color(0.9, 0.5, 1.0, 1.0);
	}
}

__global__ void rayCast(TColor *dst, int imageW, int imageH, const Scene& scene,
		const Camera& camera) {
	const int ix = blockDim.x * blockIdx.x + threadIdx.x;
	const int iy = blockDim.y * blockIdx.y + threadIdx.y;

	Radiance3 L_o;
	const Ray& R = computeEyeRay(ix + 0.5f, iy + 0.5f, imageW, imageH, camera);
	float distance = INFINITY;
	dst[imageW * iy + ix] = make_color(1.0, 1.0, 1.0, 1.0);
	for (unsigned int t = 0; t < scene.triangleCount; ++t) {
		const Triangle& T = scene.triangles[t];
		if (sampleRayTriangle(scene, ix, iy, R, T, L_o, distance)) {
			if (ix < imageW && iy < imageH) {
				dst[imageW * iy + ix] = make_color(1.0, 0.5, 0.75, 1.0);
			}
		}
	}
}

////////////////////////////////////////////////////////////////////////////////
// CUDA code handles
////////////////////////////////////////////////////////////////////////////////
extern "C" hipError_t CUDA_Bind2TextureArray() {
	return hipBindTextureToArray(texImage, a_Src);
}

extern "C" hipError_t CUDA_UnbindTexture() {
	return hipUnbindTexture(texImage);
}

extern "C" hipError_t CUDA_MallocArray(uchar4 **h_Src, int imageW,
		int imageH) {
	hipError_t error;

	error = hipMallocArray(&a_Src, &uchar4tex, imageW, imageH);
	error = hipMemcpyToArray(a_Src, 0, 0, *h_Src,
			imageW * imageH * sizeof(uchar4), hipMemcpyHostToDevice);

	return error;
}

extern "C" hipError_t CUDA_FreeArray() {
	return hipFreeArray(a_Src);
}

extern "C" void cuda_Clear(TColor *d_dst, int imageW, int imageH) {
	dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);
	dim3 grid(iDivUp(imageW, BLOCKDIM_X), iDivUp(imageH, BLOCKDIM_Y));

	Clear<<<grid, threads>>>(d_dst, imageW, imageH);
}

extern "C" void cuda_rayCasting(TColor *d_dst, int imageW, int imageH,
		const Scene& scene, const Camera& camera) {
	dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);
	dim3 grid(iDivUp(imageW, BLOCKDIM_X), iDivUp(imageH, BLOCKDIM_Y));

	rayCast<<<grid, threads>>>(d_dst, imageW, imageH, scene, camera);
}
